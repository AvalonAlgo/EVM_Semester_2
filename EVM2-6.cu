#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

using namespace std;

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    int deviceCount;
    hipDeviceProp_t deviceProp;
    //Сколько устройств CUDA установлено на PC.
    hipGetDeviceCount(&deviceCount);
    printf("Device count: %d\n\n", deviceCount);
    for (int i = 0; i < deviceCount; i++)
    {
        //Получаем информацию об устройстве
        hipGetDeviceProperties(&deviceProp, i);
        //Выводим иформацию об устройстве
        printf("Device name: %s\n", deviceProp.name);
        printf("Total global memory: %d\n",
            deviceProp.totalGlobalMem);
        printf("Shared memory per block: %d\n",
            deviceProp.sharedMemPerBlock);
        printf("Registers per block: %d\n",
            deviceProp.regsPerBlock);
        printf("Warp size: %d\n", deviceProp.warpSize);
        printf("Memory pitch: %d\n", deviceProp.memPitch);
        printf("Max threads per block: %d\n",
            deviceProp.maxThreadsPerBlock);
        printf("Max threads dimensions: x = %d, y = %d, z = %d\n",
            deviceProp.maxThreadsDim[0],
            deviceProp.maxThreadsDim[1],
            deviceProp.maxThreadsDim[2]);
        printf("Max grid size: x = %d, y = %d, z = %d\n",
            deviceProp.maxGridSize[0],
            deviceProp.maxGridSize[1],
            deviceProp.maxGridSize[2]);
        printf("Clock rate: %d\n", deviceProp.clockRate);
        printf("Total constant memory: %d\n",
            deviceProp.totalConstMem);
        printf("Compute capability: %d.%d\n", deviceProp.major,
            deviceProp.minor);
        printf("Texture alignment: %d\n",
            deviceProp.textureAlignment);
        printf("Device overlap: %d\n", deviceProp.deviceOverlap);
        printf("Multiprocessor count: %d\n",
            deviceProp.multiProcessorCount);
        printf("Kernel execution timeout enabled: %s\n",
            deviceProp.kernelExecTimeoutEnabled ? "true" : "false");
    }

    cout << endl;

    const int arraySize = 10;
    const int a[arraySize] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10 };
    const int b[arraySize] = { 10, 20, 30, 40, 50, 60, 70, 80, 90, 100 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    cout << "a + b = " << "{ ";
    for (int i = 0; i < arraySize; ++i)
    {
        cout << c[i];
        if (i < arraySize - 1)
        {
            cout << ", ";
        }
    }
    cout << " }";
    cout << endl;

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel <<<1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
